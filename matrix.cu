#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <iostream>
#include <initializer_list>
#include <cassert>
#include <algorithm>
#include <cmath>
#include <vector>
#include <numeric>
#include <random>
#include "KERNELS.cuh"
#include "matrix.cuh"

//A Wrapper Around hipMalloc For Easy And Silent Errors Free Memory Allocations
float* Matrix::allocate(int size) {
	float* d_ptr{ nullptr };
	auto alloc{ hipMalloc(&d_ptr, size * sizeof(float)) };
	CUDA_CHECK(alloc);
	return d_ptr;
}

//A Wrapper Around CudaMemcpy For Easy And Silent Errors Free Usage
void Matrix::copy(const float* dst, const float* src, int size, hipMemcpyKind kind) {
	auto cpy{ hipMemcpy((void**)dst, (void**)src, size * sizeof(float), kind) };
	CUDA_CHECK(cpy);
}

//Contructor To Construct The Object From A Float Pointer 
Matrix::Matrix(const float* data_ptr, int rows, int cols, bool destroy)
	: m_rows{ rows }, m_cols{ cols }, m_elements{ rows * cols }, m_destroy{destroy} {  //member initializer ist

	assert(m_cols != 0 && m_rows != 0 && data_ptr != nullptr); //Checks If Pointer isnt nullptr or dims arent 0
	m_data_ptr = allocate(m_elements);   //allocate memory to the m_data_ptr 
	hipPointerAttribute_t attr;
	//fetch info about the pointer
	if (hipPointerGetAttributes(&attr, data_ptr) == hipSuccess) {
		if (attr.hostPointer != nullptr) copy(m_data_ptr, data_ptr, m_elements, hipMemcpyHostToDevice); //If pointer is a host pointer
		else if (attr.devicePointer != nullptr) copy(m_data_ptr, data_ptr, m_elements, hipMemcpyDeviceToDevice); //if pointer is a device pointer
	}
}

// Constructor to initialize Matrix from a nested initializer list (like {{1,2,3},{4,5,6}})
// It flattens the 2D structure and copies the data to GPU memory
Matrix::Matrix(std::initializer_list<std::initializer_list<float>> list) {

	bool isFirst{ true };
	m_rows = list.size(); // Number of rows in the matrix

	std::vector<float> flattened; // Temporary CPU-side buffer to flatten data

	for (const auto& val : list) {
		if (isFirst) {
			m_cols = val.size();				// Infer number of columns from first row
			m_elements = m_cols * m_rows;		// Total number of elements
			flattened.reserve(m_elements);		// Reserve memory for performance
		}
		// Ensure all rows have same number of columns
		assert(val.size() == m_cols);
		// Append row elements to flat vector
		flattened.insert(flattened.end(), val.begin(), val.end());
		isFirst = false;
	}
	// Allocate memory on GPU for flattened matrix
	m_data_ptr = allocate(m_elements);
	// Copy data from CPU to GPU
	copy(m_data_ptr, flattened.data(), m_elements, hipMemcpyHostToDevice);
}

//Construct just to reserve memory 
Matrix::Matrix(int rows, int cols){
	reserve(rows, cols);
}

//Move Constructor
//Transfers ownership of resources from the temporary 'matrix' to 'this'
//Prevents deep copy and makes move operations fast and safe
Matrix::Matrix(Matrix&& matrix) noexcept 
	: m_data_ptr{ matrix.m_data_ptr },		// Take over GPU memory pointer
	m_rows{ matrix.m_rows },				// Take over row count
	m_cols{ matrix.m_cols },				// Take over col count
	m_elements{ matrix.m_elements },		// Take over total element count
	m_destroy{matrix.m_destroy} {			// Copy destruction flag (if applicable)

	//Invalidate the source matrix so it doesn't free memory we now own
	matrix.m_data_ptr = nullptr;
	matrix.m_rows = matrix.m_cols = matrix.m_elements = 0;
	
}

// Copy Constructor
// Delegates to another constructor that accepts raw data pointer and dimensions
Matrix::Matrix(Matrix& matrix) : Matrix{ matrix.m_data_ptr, matrix.m_cols, matrix.m_rows, matrix.m_destroy } {}

// Reserves GPU memory for the matrix with given dimensions
// Frees and reallocates only if the size changes
void Matrix::reserve(int rows, int cols) {
	int elements{ rows * cols };

	// Reallocate only if new size differs from current size
	if (m_elements != elements) {
		hipFree(m_data_ptr);				// Free existing GPU memory
		m_data_ptr = allocate(elements);	// Allocate new memory on GPU
	}
	// Update metadata
	m_rows = rows; m_cols = cols; m_elements = elements;
}

// Move assignment operator
void Matrix::operator=(Matrix&& matrix) noexcept {
	// Handle self-assignment
	if (this == &matrix) return;
	// Free existing GPU memory, if allocated
	if (m_data_ptr) hipFree(m_data_ptr);

	// Transfer ownership of data and metadata from the source matrix
	m_data_ptr = matrix.m_data_ptr;
	m_cols = matrix.m_cols;
	m_rows = matrix.m_rows;
	m_elements = matrix.m_elements;
	m_destroy = matrix.m_destroy;

	// Nullify the source matrix to avoid double free or accidental use
	matrix.m_data_ptr = nullptr;
	matrix.m_rows = matrix.m_cols = matrix.m_elements = 0; // Prevent destruction on moved-from object (optional safety)
}

// Clones the contents and structure of another Matrix into this one
void Matrix::clone(Matrix& matrix) {
	// If element count differs, free old memory and allocate new space
	if (matrix.m_elements != m_elements) {
		hipFree(m_data_ptr);							// Free current memory
		m_data_ptr = allocate(matrix.m_elements);		// Allocate new memory matching the source matrix
	}	

	// Copy metadata
	m_elements = matrix.m_elements;
	m_rows = matrix.m_rows;
	m_cols = matrix.m_cols;
	m_destroy = matrix.m_destroy;

	// Copy data from source matrix on device to this matrix on device (device-to-device copy)
	copy(m_data_ptr, matrix.m_data_ptr, matrix.m_elements, hipMemcpyDeviceToDevice);
}

// Extracts a block of consecutive rows from the matrix.
// If `isView` is true, it returns a non-owning view (no memory copied)
// If `isView` is false, it returns a deep copy of the rows (memory is allocated and data is copied)
Matrix Matrix::rowBlock(int from, int rows, bool isView){
	Matrix mat{};  // Resultant sub-matrix

	if (isView) {
		// Return a view: just adjust pointer and metadata, no allocation or copying
		mat.m_data_ptr = m_data_ptr+(m_cols*from);  // Point to the correct offset in original data
		mat.m_rows = rows;			
		mat.m_cols = m_cols;
		mat.m_elements = rows * m_cols;
		mat.m_destroy = false;  // Don't free memory, since it’s just a view
		return mat;
	}
	else
	{
		// Return a copy: allocate new memory and copy the data
		mat.m_rows = rows;
		mat.m_cols = m_cols;
		mat.m_elements = rows * m_cols;
		mat.m_data_ptr = allocate(mat.m_elements);  // Allocate new GPU memory
		copy(mat.m_data_ptr, m_data_ptr + (m_cols * from), sizeof(float)*mat.m_elements, hipMemcpyDeviceToDevice);
		return mat;
	}
}

// Moves data ownership from an existing device pointer into this Matrix object.
// Assumes the pointer is already allocated on the device and valid.
// This function does not copy data, it just takes ownership (performs a move)
void Matrix::moveFrom(float* device_ptr, int rows, int cols) {

	hipPointerAttribute_t attr;
	hipPointerGetAttributes(&attr, device_ptr); // Get pointer attributes to ensure it's a device pointer
	assert(attr.devicePointer != nullptr);		 //must be a valid device pointer

	int ptr_size{ rows * cols};
	hipFree(m_data_ptr);				// Free current device memory

	// Update metadata
	m_rows = rows; m_cols = cols; m_elements = ptr_size;
	// Take ownership of the incoming device memory
	m_data_ptr = device_ptr;
	// Invalidate the source device pointer
	device_ptr = nullptr;
}

// Shuffles the current matrix's rows using the provided device-side indices,
// and stores the result in another pre-allocated matrix.
// Parameters:
// - matrix: the output Matrix object where shuffled data will be stored
// - device_indices: device pointer holding shuffled row indices
// - free_indices_after: whether to free device_indices after use
void Matrix::shuffle(Matrix& matrix, int* device_indices, bool free_indices_after) {
	assert(this!=&matrix); // Ensure source and destination are not the same matrix

	// Define CUDA block and grid dimensions for 2D execution
	dim3 BLOCK_SIZE(16, 16);
	dim3 GRID_SIZE(
		(ceil(m_cols / static_cast<float>(BLOCK_SIZE.x))),
		(ceil(m_rows / static_cast<float>(BLOCK_SIZE.y)))
	);

	// Launch the shuffle kernel on the device
	Kernels::shuffle<<<GRID_SIZE, BLOCK_SIZE>>>(m_data_ptr, matrix.m_data_ptr, device_indices, m_rows, m_cols);
	// Wait for GPU to finish and check for any kernel errors
	hipDeviceSynchronize();
	CUDA_CHECK(hipGetLastError());
	// Optionally free the device-side indices after shuffling
	if (free_indices_after) hipFree(device_indices);
}

// Transposes the current matrix and stores the result in a given destination matrix
// The destination matrix must already be allocated with the correct number of elements
void Matrix::transpose(Matrix& matrix) const {
	assert(m_elements == matrix.m_elements && this != &matrix); // Ensure same size and not transposing into self

	// Update the shape of the destination matrix: rows become columns and vice versa
	matrix.m_cols = m_rows;
	matrix.m_rows = m_cols;

	// Define CUDA block and grid dimensions for 2D execution
	dim3 BLOCK_SIZE(16, 16);
	dim3 GRID_SIZE(
		(ceil(m_cols / static_cast<float>(BLOCK_SIZE.x))),
		(ceil(m_rows / static_cast<float>(BLOCK_SIZE.y)))
	);

	// Launch the CUDA transpose kernel
	Kernels::transpose << <GRID_SIZE, BLOCK_SIZE >> > (m_data_ptr,
		matrix.m_data_ptr, m_rows, m_cols);
	// Wait for GPU to finish and check for any kernel errors
	hipDeviceSynchronize();
	CUDA_CHECK(hipGetLastError());
}

//Prints the matrix to console
void print(const Matrix& matrix) {
	//Launch the kernel to print the matrix
	Kernels::print_data << <1, 1 >> > (matrix.m_data_ptr, matrix.m_rows, matrix.m_cols);
	hipDeviceSynchronize();
	CUDA_CHECK(hipGetLastError());
}

// Adds current matrix to another matrix and stores result in store_in
void Matrix::addInto(Matrix& matrix, Matrix& store_in){
	assert(matrix.m_cols == m_cols && matrix.m_rows == m_rows
		&& store_in.m_cols == m_cols && store_in.m_rows == m_rows); // Ensure dimensions match
	
	// Define CUDA block and grid dimensions
	dim3 BLOCK_SIZE(16, 16);
	dim3 GRID_SIZE(
		(ceil(m_cols / static_cast<float>(BLOCK_SIZE.x))),
		(ceil(m_rows / static_cast<float>(BLOCK_SIZE.y)))
	);

	// Launch addition kernel (operationID = 0 for addition)
	Kernels::matrixOperations << <GRID_SIZE, BLOCK_SIZE >> > (m_data_ptr, matrix.m_data_ptr, 
		store_in.m_data_ptr, m_rows, m_cols, 0);
	CUDA_CHECK(hipGetLastError());
}

// Subtracts another matrix from this matrix and stores result in store_in
void Matrix::subtractInto(Matrix& matrix, Matrix& store_in) {
	assert(matrix.m_cols == m_cols && matrix.m_rows == m_rows
		&& store_in.m_cols == m_cols && store_in.m_rows == m_rows); // Ensure dimensions match

	// Define CUDA block and grid dimensions
	dim3 BLOCK_SIZE(16, 16);
	dim3 GRID_SIZE(
		(ceil(m_cols / static_cast<float>(BLOCK_SIZE.x))),
		(ceil(m_rows / static_cast<float>(BLOCK_SIZE.y)))
	);

	// Launch subtraction kernel (operationID = 1 for subtraction)
	Kernels::matrixOperations << <GRID_SIZE, BLOCK_SIZE >> > (m_data_ptr, matrix.m_data_ptr,
		store_in.m_data_ptr, m_rows, m_cols, 1);
	hipDeviceSynchronize();
	CUDA_CHECK(hipGetLastError());
}

// Performs matrix multiplication: this * mat -> store_in
void Matrix::MatMulInto(Matrix& mat, Matrix& store_in){
	assert(store_in.m_rows == m_rows && store_in.m_cols == mat.m_cols); // Ensure output shape is correct
	assert(m_cols == mat.m_rows); // Ensure inner dimensions match for multiplication

	// Define CUDA block and grid dimensions
	dim3 BLOCK_SIZE(16, 16);
	dim3 GRID_SIZE(
		(ceil(mat.m_cols / static_cast<float>(BLOCK_SIZE.x))),
		(ceil(m_rows / static_cast<float>(BLOCK_SIZE.y)))
	);

	// Launch CUDA matmul kernel
	Kernels::matmul << <GRID_SIZE, BLOCK_SIZE >> > (m_data_ptr, mat.data(), store_in.data(),
		m_rows, mat.m_cols, m_cols);
	hipDeviceSynchronize();
	CUDA_CHECK(hipGetLastError());
}

//Set all the elements of Matrix to the specified constant
void Matrix::setConstant(float constant) {
	// Define CUDA block and grid dimensions
	dim3 BLOCK_SIZE(16, 16);
	dim3 GRID_SIZE(
		(ceil(m_cols / static_cast<float>(BLOCK_SIZE.x))),
		(ceil(m_rows / static_cast<float>(BLOCK_SIZE.y)))
	);
	// Launch CUDA constant kernel
	Kernels::constant << <GRID_SIZE, BLOCK_SIZE >> > (m_data_ptr, constant, m_rows, m_cols);
	hipDeviceSynchronize();
	CUDA_CHECK(hipGetLastError());
}

// Performs element-wise multiplication b/w this and passed matrix and store the result in store_in
void Matrix::elementWiseProduct(Matrix& matrix, Matrix& store_in) {
	assert(store_in.m_rows == m_rows && store_in.m_cols == m_cols);  // Check output shape
	assert(matrix.m_rows == m_rows && matrix.m_cols == m_cols);		// Ensure input matrices match in size

	// Define CUDA block and grid dimensions
	dim3 BLOCK_SIZE(16, 16);
	dim3 GRID_SIZE(
		(ceil(m_cols / static_cast<float>(BLOCK_SIZE.x))),
		(ceil(m_rows / static_cast<float>(BLOCK_SIZE.y)))
	);

	// Launch CUDA kernel for element-wise product
	Kernels::elementWiseProduct << <GRID_SIZE, BLOCK_SIZE >> > (m_data_ptr, matrix.m_data_ptr, store_in.m_data_ptr, m_rows, m_cols);
	hipDeviceSynchronize();
	CUDA_CHECK(hipGetLastError());
}

// Adds a row vector to each row of the matrix (broadcasting across rows)
void Matrix::rowBroadcast(Matrix& vector, Matrix& store_in) {
	assert(vector.m_cols == m_cols && vector.m_rows == 1     // Ensure vector is 1xN
		&& store_in.m_rows==m_rows && store_in.m_cols==m_cols);  // Ensure output shape matches this matrix
	
	// Define CUDA block and grid dimensions
	dim3 BLOCK_SIZE(16, 16);
	dim3 GRID_SIZE(
		(ceil(m_cols / static_cast<float>(BLOCK_SIZE.x))),
		(ceil(m_rows / static_cast<float>(BLOCK_SIZE.y)))
	);
	// Launch CUDA kernel to perform row-wise broadcasting
	Kernels::rowBroadcast << <GRID_SIZE, BLOCK_SIZE >> > (vector.data(), m_data_ptr,
		store_in.data(), m_rows, m_cols);

	hipDeviceSynchronize();
	CUDA_CHECK(hipGetLastError());
}

// Computes the derivative (gradient) of the activation function element-wise
// and stores the result in 'store_in'.
void Matrix::activationPrimes(Matrix& store_in, const std::string& activation_fn) {
	assert(store_in.m_rows == m_rows && store_in.m_cols == m_cols);  // Shapes must match
	// Define CUDA block and grid dimensions
	dim3 BLOCK_SIZE(16, 16);
	dim3 GRID_SIZE(
		(ceil(m_cols / static_cast<float>(BLOCK_SIZE.x))),
		(ceil(m_rows / static_cast<float>(BLOCK_SIZE.y)))
	);
	//sigmoid = 0, relu = 1, leaky_relu = 2, tanh = 3
	int activation_id{};
	if (activation_fn == "sigmoid") activation_id = 0;
	else if (activation_fn == "relu") activation_id = 1;
	else if (activation_fn == "leaky_relu") activation_id = 2;
	else if (activation_fn == "tanh") activation_id = 3;
	else if (activation_fn == "linear") {
		store_in.setConstant(1.0f);
		return;
	}
	// Launch CUDA kernel to compute the element-wise activation prime
	Kernels::activationPrime << <GRID_SIZE, BLOCK_SIZE >> > (m_data_ptr, store_in.m_data_ptr, activation_id, m_rows, m_cols);
	hipDeviceSynchronize();
	CUDA_CHECK(hipGetLastError());

}

// Broadcasts a column vector (1D) across each column of the matrix and stores result in 'store_in'
// Each element in a row gets the corresponding vector value added to it
void Matrix::colBroadcast(Matrix& vector, Matrix& store_in) {
	assert(vector.m_rows == m_rows && vector.m_cols == 1    // Vector should have same row count
		&& store_in.m_rows == m_rows && store_in.m_cols == m_cols);    // Output matrix shape must match

	// Define CUDA block and grid dimensions
	dim3 BLOCK_SIZE(16, 16);
	dim3 GRID_SIZE(
		(ceil(m_cols / static_cast<float>(BLOCK_SIZE.x))),
		(ceil(m_rows / static_cast<float>(BLOCK_SIZE.y)))
	);

	// Launch kernel to add the broadcasted column vector to each column of the matrix
	Kernels::colBroadcast << <GRID_SIZE, BLOCK_SIZE >> > (vector.data(), m_data_ptr,
		store_in.data(), m_rows, m_cols);

	hipDeviceSynchronize();
	CUDA_CHECK(hipGetLastError());
}

//Add Up All the Columns Vertically
void Matrix::rowWiseSum(Matrix& store_in) {
	assert(store_in.m_cols == m_cols && store_in.m_rows==1); //store_in must be of dims (1, N)

	//Define block size and threads per block
	int THREADS{ 256 };
	int BLOCK_SIZE = ceil(m_cols / static_cast<float>(THREADS));
	
	//launch kernel
	Kernels::rowWiseSum << <BLOCK_SIZE, THREADS >> > (m_data_ptr, store_in.m_data_ptr, m_rows, m_cols);
	hipDeviceSynchronize();
	CUDA_CHECK(hipGetLastError());
}

// Applies a scalar operation element-wise to the matrix and stores the result in 'store_in'.
// Supported operations: '+', '-', '*', '/'.
void Matrix::scalarOperation(float scalar, const char& operation, Matrix& store_in) { 
	 
	assert(store_in.m_rows == m_rows && store_in.m_cols == m_cols);  // Ensure dimensions match

	dim3 BLOCK_SIZE(16, 16);
	dim3 GRID_SIZE(
		(ceil(m_cols / static_cast<float>(BLOCK_SIZE.x))),
		(ceil(m_rows / static_cast<float>(BLOCK_SIZE.y)))
	);

	int operation_id{};

	// Map operation character to operation ID
	if (operation == '+') operation_id = 0;
	else if (operation == '-') operation_id = 1;
	else if (operation == '/') operation_id = 2;
	else if (operation == '*') operation_id = 3;
	else throw "Invalid Operation";  // Throw error for unsupported ops

	// Launch CUDA kernel to perform the scalar operation
	Kernels::scalarOperations << <GRID_SIZE, BLOCK_SIZE >> > (m_data_ptr, store_in.m_data_ptr, scalar, m_rows, m_cols, operation_id);
	hipDeviceSynchronize();
	CUDA_CHECK(hipGetLastError());
}

//Apply relu activation on the matrix
void Matrix::relu(Matrix& store) {
	assert(store.m_rows == m_rows && store.m_cols == m_cols); //check the opt matrix dims

	//Configure grid and block sizes
	dim3 BLOCK_SIZE(16, 16);
	dim3 GRID_SIZE(
		(ceil(m_cols / static_cast<float>(BLOCK_SIZE.x))),
		(ceil(m_rows / static_cast<float>(BLOCK_SIZE.y)))
	);

	//launch kernel to apply relu
	Kernels::relu<<<GRID_SIZE, BLOCK_SIZE>>>(m_data_ptr, store.m_data_ptr, m_rows, m_cols);
	hipDeviceSynchronize();
	CUDA_CHECK(hipGetLastError());
}

//Apply sigmoid activation on the matrix
void Matrix::sigmoid(Matrix& store) {
	assert(store.m_rows == m_rows && store.m_cols == m_cols); //check the opt matrix dims

	//Configure grid and block sizes
	dim3 BLOCK_SIZE(16, 16);
	dim3 GRID_SIZE(
		(ceil(m_cols / static_cast<float>(BLOCK_SIZE.x))),
		(ceil(m_rows / static_cast<float>(BLOCK_SIZE.y)))
	);

	//launch kernel to apply sigmoid
	Kernels::sigmoid << <GRID_SIZE, BLOCK_SIZE >> > (m_data_ptr, store.m_data_ptr, m_rows, m_cols);
	hipDeviceSynchronize();
	CUDA_CHECK(hipGetLastError());
}

//Apply leaky_relu activation on the matrix
void Matrix::leaky_relu(Matrix& store) {
	assert(store.m_rows == m_rows && store.m_cols == m_cols); //check the opt matrix dims

	//Configure grid and block sizes
	dim3 BLOCK_SIZE(16, 16);
	dim3 GRID_SIZE(
		(ceil(m_cols / static_cast<float>(BLOCK_SIZE.x))),
		(ceil(m_rows / static_cast<float>(BLOCK_SIZE.y)))
	);

	//launch kernel to apply leaky_relu
	Kernels::leaky_relu << <GRID_SIZE, BLOCK_SIZE >> > (m_data_ptr, store.m_data_ptr, m_rows, m_cols);
	hipDeviceSynchronize();
	CUDA_CHECK(hipGetLastError());
}

//Apply tanh activation on the matrix
void Matrix::tanh(Matrix& store) {
	assert(store.m_rows == m_rows && store.m_cols == m_cols); //check the opt matrix dims

	//Configure grid and block sizes
	dim3 BLOCK_SIZE(16, 16);
	dim3 GRID_SIZE(
		(ceil(m_cols / static_cast<float>(BLOCK_SIZE.x))),
		(ceil(m_rows / static_cast<float>(BLOCK_SIZE.y)))
	);
	//launch kernel to apply tanh
	Kernels::tanh << <GRID_SIZE, BLOCK_SIZE >> > (m_data_ptr, store.m_data_ptr, m_rows, m_cols);
	hipDeviceSynchronize();
	CUDA_CHECK(hipGetLastError());
}

// Applies softmax activation across each row of the matrix and stores the result in 'store'
void Matrix::softmax(Matrix& store) { 
	assert(store.m_rows == m_rows && store.m_cols == m_cols); // Ensure output dimensions match
	
	// Allocate temporary GPU memory for row-wise max logits and exp sums
	float* max_logit{allocate(m_rows)};
	float* exp_sum{allocate(m_rows)};

	// Launch 1D kernel to compute max logits and exp sums for numerical stability
	int BLOCK_SIZE_{ 256 };
	int GRID_SIZE_ = (m_rows + BLOCK_SIZE_ - 1) / BLOCK_SIZE_;

	Kernels::calculate_max_and_sum_exp << <GRID_SIZE_, BLOCK_SIZE_ >> > (m_data_ptr,
		exp_sum, max_logit, m_rows, m_cols);
	
	hipDeviceSynchronize();
	CUDA_CHECK(hipGetLastError());

	// Launch 2D kernel to compute final softmax values
	dim3 BLOCK_SIZE(16, 16);
	dim3 GRID_SIZE(
		(ceil(m_cols / static_cast<float>(BLOCK_SIZE.x))),
		(ceil(m_rows / static_cast<float>(BLOCK_SIZE.y)))
	);

	Kernels::softmax << <GRID_SIZE, BLOCK_SIZE >> > (m_data_ptr, store.data(),
		exp_sum, max_logit, m_rows, m_cols);

	hipDeviceSynchronize();
	CUDA_CHECK(hipGetLastError());

	// Free temporary buffers
	hipFree(max_logit);
	hipFree(exp_sum);
}

//reset the dimensions of the matrix
void Matrix::setDims(int rows, int cols) {
	assert(rows * cols <= m_elements); //row * col should be less or equal to the m_elements
	m_rows = rows;
	m_cols = cols;
}

// Generates a shuffled array of indices on the host and transfers it to the GPU.
// - seed: for deterministic shuffling
// - rows: number of indices to generate (0 to rows-1)
// Returns a device pointer to the shuffled indices array.
int* make_device_indices(unsigned int seed, int rows) {
	std::vector<int> indices(rows); // Host vector to hold indices
	// Allocate device memory
	int* d_indices;
	auto alloc{ hipMalloc(&d_indices, sizeof(int) * indices.size()) };
	CUDA_CHECK(alloc);
	// Fill with sequential values [0, 1, 2, ..., rows-1]
	std::iota(indices.begin(), indices.end(), 0);
	// Shuffle using given seed for reproducibility
	std::shuffle(indices.begin(), indices.end(), std::mt19937(seed));

	// Copy shuffled indices from host to device
	auto cpy{ hipMemcpy((void**)d_indices, (void**)indices.data(),
	sizeof(int) * indices.size(), hipMemcpyHostToDevice) };
	CUDA_CHECK(cpy);

	return d_indices; // Return device pointer
}

// Generates a shuffled array of indices on the host and transfers it to the GPU.
// returns a device pointer to the shuffled indices array, but doesnt require a seed argument
int* make_device_indices(int rows) {
	std::vector<int> indices(rows);  // host vector to hold indices
	int* d_indices;  // Allocate device memory
	auto alloc{ hipMalloc(&d_indices, sizeof(int) * indices.size()) };
	CUDA_CHECK(alloc);

	std::random_device rd;  //seed

	std::iota(indices.begin(), indices.end(), 0);  // Fill with sequential values [0, 1, 2, ..., rows-1]
	std::shuffle(indices.begin(), indices.end(), std::mt19937(rd())); // Shuffle using given seed for reproducibility
	// Copy shuffled indices from host to device
	auto cpy{ hipMemcpy((void**)d_indices, (void**)indices.data(),
	sizeof(int) * indices.size(), hipMemcpyHostToDevice) };
	CUDA_CHECK(cpy);

	return d_indices;
}

// Launches CUDA kernel to compute dC/da (partial derivative of cost w.r.t. activations)
// depending on the specified loss function
// Supports: MSE (0), Binary Cross-Entropy (1), Cross-Entropy (2)
void compute_dC_da_GPU(Matrix activations, Matrix Y_train, Matrix errors, const std::string& loss_function) {

	int lossFn_id{};
	if (loss_function == "MSE") lossFn_id = 0;
	else if (loss_function == "binary_cross_entropy") lossFn_id = 1;
	else if (loss_function == "cross_entropy") lossFn_id = 2;
	
	// Configure CUDA thread block and grid dimensions
	dim3 BLOCK_SIZE(16, 16);
	dim3 GRID_SIZE(
		(ceil(errors.getCols() / static_cast<float>(BLOCK_SIZE.x))),
		(ceil(errors.getRows() / static_cast<float>(BLOCK_SIZE.y)))
	);

	// Launch CUDA kernel to compute dC/da
	Kernels::compute_dC_da << <GRID_SIZE, BLOCK_SIZE >> > (activations.data(), 
		Y_train.data(), errors.data(), lossFn_id, errors.getRows(), errors.getCols());
	hipDeviceSynchronize();
	CUDA_CHECK(hipGetLastError());
}

//Calculate Cross Entropy Loss
void cross_entropy_GPU(Matrix activations, Matrix Y_train, float* d_loss_ptr) {

	//Define blocks and threads per block
	int THREADS = 256;
	int BLOCKS = std::ceil(activations.getRows() / static_cast<float>(THREADS));
	//launch the Cross Entropy Kernel
	Kernels::cross_entropy_loss << <BLOCKS, THREADS >> > (activations.data(), Y_train.data(),
		d_loss_ptr, activations.getRows(), activations.getCols());
	hipDeviceSynchronize();
	CUDA_CHECK(hipGetLastError());
}

//Calculate Binary Cross Entropy Loss
void binary_cross_entropy_GPU(Matrix activations, Matrix Y_train, float* d_loss_ptr) {
	//Define blocks and threads per block
	int THREADS = 256;
	int BLOCKS = std::ceil(activations.getRows() / static_cast<float>(THREADS));
	//launch the Binary Cross Entropy Kernel
	Kernels::binary_cross_entropy << <BLOCKS, THREADS >> > (activations.data(), Y_train.data(),
		d_loss_ptr, activations.getRows(), activations.getCols());
	hipDeviceSynchronize();
	CUDA_CHECK(hipGetLastError());
}

//Calculate MSE Loss
void MSE_GPU(Matrix activations, Matrix Y_train, float* d_loss_ptr) {
	//Define blocks and threads per block
	int THREADS = 256;
	int BLOCKS = std::ceil(activations.getRows() / static_cast<float>(THREADS));
	//launch the MSE Kernel
	Kernels::MSE << <BLOCKS, THREADS >> > (activations.data(), Y_train.data(),
		d_loss_ptr, activations.getRows(), activations.getCols());
	hipDeviceSynchronize();
	CUDA_CHECK(hipGetLastError());
}
