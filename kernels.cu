#include "hip/hip_runtime.h"
﻿#include "KERNELS.cuh"
#include <cstdio>
#include <cmath>

//CUDA Kernel To Print "Matrix" Objects
__global__ void Kernels::print_data(float* data, int rows, int cols) {

	printf("Matrix( [");
	for (int row{}; row < rows; ++row) {

		if (row > 0) printf("%9s", "");
		for (int col{}; col < cols; ++col) {
			int idx{ row * cols + col };

			printf("%f", data[idx]);

			if (row == (rows - 1) && col == (cols - 1)) printf("");
			else printf(", ");
		}
		if (row != rows - 1) printf("\n");
	}
	printf("] )\n");
}

// CUDA kernel for performing element-wise matrix operations on two matrices
// Supported operations:
// operationID == 0 -> Element-wise addition: matrix_c = matrix_a + matrix_b
// operationID == 1 -> Element-wise subtraction: matrix_c = matrix_a - matrix_b
__global__ void Kernels::matrixOperations(float* matrix_a, float* matrix_b, float* matrix_c, int rows, int cols, int operationID) {
	int row = blockDim.y * blockIdx.y + threadIdx.y; // Calculate row index
	int col = blockDim.x * blockIdx.x + threadIdx.x; // Calculate col index
	int idx = row * cols + col; // Flattened index for 2D array

	// Ensure thread is within matrix bounds
	if (row < rows && col < cols) {
		if(operationID==0) matrix_c[idx] = matrix_a[idx] + matrix_b[idx]; // Element-wise addition
		else if(operationID==1) matrix_c[idx] = matrix_a[idx] - matrix_b[idx]; // Element-wise subtraction
	}
}

// CUDA kernel for performing element-wise scalar operations on a matrix
// mat_b = mat_a (op) scalar
// Supported operations based on operationID:
// 0 -> Addition:       mat_b = mat_a + scalar
// 1 -> Subtraction:    mat_b = mat_a - scalar
// 2 -> Division:       mat_b = mat_a / scalar
// 3 -> Multiplication: mat_b = mat_a * scalar
__global__ void Kernels::scalarOperations(float* mat_a, float* mat_b, float scalar, int rows, int cols, int operationID) {
	int row = blockDim.y * blockIdx.y + threadIdx.y; // Calculate row index
	int col = blockDim.x * blockIdx.x + threadIdx.x; // Calculate col index
	int  idx = row * cols + col; // Flattened index for 2D array

	// Make sure the thread is within bounds
	if (row < rows && col < cols) {
		if (operationID==0) mat_b[idx] = mat_a[idx] + scalar; //Addition
		else if (operationID==1) mat_b[idx] = mat_a[idx] - scalar; //Substraction
		else if (operationID == 2) mat_b[idx] = mat_a[idx] / scalar; //Division
		else if (operationID==3) mat_b[idx] = mat_a[idx] * scalar;	//Multiplication
	}
}

// CUDA kernel to compute the transpose of a matrix
// Input:
// - matrix     : Pointer to the original matrix (row-major)
// - matrix_t   : Pointer to the transposed matrix output
// - rows       : Number of rows in the original matrix
// - cols       : Number of columns in the original matrix
//
// Each thread handles one element: it reads (row, col) from the input matrix
// and writes it to (col, row) in the transposed output
__global__ void Kernels::transpose(float* matrix, float* matrix_t, int rows, int cols) {
	int row = blockDim.y * blockIdx.y + threadIdx.y;  // Compute row index
	int col = blockDim.x * blockIdx.x + threadIdx.x;  // Compute col index
	int idx_nt = row * cols + col;	// Index in the original matrix
	int idx_t = col * rows + row;	// Index in the transposed matrix

	if (row < rows && col < cols) {
		matrix_t[idx_t] = matrix[idx_nt];  // Write transposed value
	}
}

// CUDA kernel to shuffle the rows of a matrix based on a provided index mapping
// Input:
// - original : Pointer to the original matrix (row-major order)
// - shuffled : Pointer to the output matrix where shuffled data will be stored
// - indices  : Pointer to an array of shuffled row indices
// - rows     : Number of rows in the matrix
// - cols     : Number of columns in the matrix
//
// Each thread copies one element from the original matrix to the shuffled matrix
// according to the shuffled row index given in 'indices'

__global__ void Kernels::shuffle(float* original, float* shuffled, int* indices, int rows, int cols) {
	int rowID = blockDim.y * blockIdx.y + threadIdx.y; // Compute row index
	int colID = blockDim.x * blockIdx.x + threadIdx.x; // Compute col index
	if (rowID < rows && colID < cols) {
		// Copy the element from the original matrix based on shuffled row index
		shuffled[rowID * cols + colID] = original[indices[rowID] * cols + colID];
	}
}

// CUDA kernel to set all elements of a matrix to a constant value
// Input:
// - mat      : Pointer to the matrix in device memory (row-major order)
// - constant : The constant value to assign to each element
// - rows     : Number of rows in the matrix
// - cols     : Number of columns in the matrix
//
// Each thread sets one element of the matrix to the specified constant
__global__ void Kernels::constant(float* mat, float constant, int rows, int cols) {
	int row = blockDim.y * blockIdx.y + threadIdx.y; // Row index for current thread
	int col = blockDim.x * blockIdx.x + threadIdx.x; // Column index for current thread
	
	if (row < rows && col < cols) {
		mat[row * cols + col] = constant; // Set matrix element to constant
	}
}

// CUDA kernel to perform row-wise broadcasting of a vector over a matrix
// It adds the corresponding element from the vector to each element in the matrix row-wise
// 
// Input:
// - vec      : Pointer to the 1D vector of size [cols], stored in device memory
// - mat      : Pointer to the 2D matrix [rows x cols], stored in row-major format
// - store_in : Pointer to the output matrix of the same size as `mat`
// - rows     : Number of rows in the matrix
// - cols     : Number of columns in the matrix
//
// Each thread handles one element in the output matrix.
__global__ void Kernels::rowBroadcast(float* vec, float* mat, float* store_in, int rows, int cols) {
	int row = blockIdx.y * blockDim.y + threadIdx.y; //Row Index
	int col = blockIdx.x * blockDim.x + threadIdx.x; //Col Index
	if (row<rows && col<cols) {
		// Add corresponding vector element to each matrix element (broadcasting)
		store_in[row * cols + col] =  mat[row*cols+col] + vec[col];
	}
}

// CUDA kernel to perform column-wise broadcasting of a vector over a matrix
// It adds the corresponding element from the vector to each element in the matrix column-wise.
//
// Input:
// - vec      : Pointer to a 1D vector of size [rows], stored in device memory
// - mat      : Pointer to the 2D matrix [rows x cols], stored in row-major format
// - store_in : Pointer to the output matrix of the same size as `mat`
// - rows     : Number of rows in the matrix
// - cols     : Number of columns in the matrix
//
// Each thread handles one element in the output matrix

__global__ void Kernels::colBroadcast(float* vec, float* mat, float* store_in, int rows, int cols) {
	int row = blockIdx.y * blockDim.y + threadIdx.y; //Row Index
	int col = blockIdx.x * blockDim.x + threadIdx.x; //Col Index
	if (row < rows && col < cols) {
		// Add the row-specific vector value to each column element in that row
		store_in[row * cols + col] = mat[row * cols + col] + vec[row];
	}
}

// CUDA kernel for matrix multiplication: C = A * B
// A: [M x K], B: [K x N], C: [M x N]
// All matrices are in row-major order
//
// Each thread computes one element of matrix C by taking the dot product of
// one row of A and one column of B.
//
// Parameters:
// - A: Input matrix A of size M x K
// - B: Input matrix B of size K x N
// - C: Output matrix C of size M x N
// - M: Number of rows in matrix A (and C)
// - N: Number of columns in matrix B (and C)
// - K: Number of columns in A and rows in B

__global__ void Kernels::matmul(const float* A, const float* B, float* C, int M, int N, int K) {

	int row = blockIdx.y * blockDim.y + threadIdx.y;// Row index of C
	int col = blockIdx.x * blockDim.x + threadIdx.x; // Row index of C

	if (row < M && col < N) {
		float sum = 0.0f;
		for (int k = 0; k < K; ++k) {
			sum += A[row * K + k] * B[k * N + col]; // Dot product
		}
		C[row * N + col] = sum;
	}
}

// CUDA kernel to apply ReLU activation function element-wise
// ReLU(x) = max(0, x)
// 
// Parameters:
// - preActivations: Input matrix of pre-activation values [rows x cols]
// - activations: Output matrix to store activated values [rows x cols]
// - rows: Number of rows in the input/output matrices
// - cols: Number of columns in the input/output matrices
//
// Each thread computes the ReLU for one element

__global__ void Kernels::relu(float* preActivations, float* activations, int rows, int cols) {
	int row = blockDim.y * blockIdx.y + threadIdx.y; //Row Index
	int col = blockDim.x * blockIdx.x + threadIdx.x; //Col Index

	if (row < rows && col < cols) {
		activations[row * cols + col] = std::fmaxf(0.0f, preActivations[row * cols + col]); //apply relu
	}
}

// CUDA kernel to apply the Sigmoid activation function element-wise.
// Sigmoid(x) = 1 / (1 + exp(-x))
//
// Parameters:
// - preActivations: Input matrix of pre-activation values [rows x cols]
// - activations: Output matrix to store activated values [rows x cols]
// - rows: Number of rows in the input/output matrices
// - cols: Number of columns in the input/output matrices
//
// Each thread computes the Sigmoid activation for one element.
__global__ void Kernels::sigmoid(float* preActivations, float* activations, int rows, int cols) {
	int row = blockDim.y * blockIdx.y + threadIdx.y;//Row Index
	int col = blockDim.x * blockIdx.x + threadIdx.x;//Col Index

	if (row < rows && col < cols) {
		activations[row * cols + col] = 1.0f / (1.0f + std::expf(-preActivations[row * cols + col])); //apply sigmoid
	}
}

// CUDA kernel to apply the Leaky ReLU activation function element-wise
// Leaky ReLU(x) = x        if x > 0
//                 0.1 * x  if x <= 0
//
// Parameters:
// - preActivations: Input matrix of pre-activation values [rows x cols]
// - activations: Output matrix to store activated values [rows x cols]
// - rows: Number of rows in the input/output matrices
// - cols: Number of columns in the input/output matrices
//
// Each thread processes one element and applies the Leaky ReLU activation
__global__ void Kernels::leaky_relu(float* preActivations, float* activations, int rows, int cols) {
	int row = blockDim.y * blockIdx.y + threadIdx.y;//Row Index
	int col = blockDim.x * blockIdx.x + threadIdx.x;//Col Index
	int idx = row * cols + col;

	if (row < rows && col < cols) {
		activations[idx] = preActivations[idx] > 0.0f ? preActivations[idx] : 0.1f * preActivations[idx]; //apply leaky relu
	}
}

// CUDA kernel to apply the Tanh activation function element-wise
// Tanh(x) = (e^x - e^(-x)) / (e^x + e^(-x))
// It squashes input values to the range [-1, 1]
//
// Parameters:
// - preActivations: Input matrix of pre-activation values [rows x cols]
// - activations: Output matrix to store activated values [rows x cols]
// - rows: Number of rows in the input/output matrices
// - cols: Number of columns in the input/output matrices
//
// Each thread computes the Tanh activation for one element
__global__ void Kernels::tanh(float* preActivations, float* activations, int rows, int cols) {
	int row = blockDim.y * blockIdx.y + threadIdx.y;//Row Index
	int col = blockDim.x * blockIdx.x + threadIdx.x;//Col Index
	int idx = row * cols + col;

	if (row < rows && col < cols) {
		activations[idx] = std::tanhf(preActivations[idx]); //apply tanh
	}
}

// CUDA kernel to compute the exponential (e^x) of each element in the input matrix
//
// Parameters:
// - activations: Input matrix [rows x cols], contains the values to exponentiate
// - store_in: Output matrix [rows x cols], where the exponentiated values will be stored
// - rows: Number of rows in the input/output matrices
// - cols: Number of columns in the input/output matrices
//
// Each thread computes the exponential of a single element using std::expf for float precision
__global__ void Kernels::calculate_exps(float* activations, float* store_in, int rows, int cols) {
	int row = blockDim.y * blockIdx.y + threadIdx.y;//Row Index
	int col = blockDim.x * blockIdx.x + threadIdx.x;//Col Index
	int idx = row * cols + col;

	if (row < rows && col < cols) {
		store_in[idx] = std::expf(activations[idx]); 
	}
}

// CUDA kernel to compute the max and the sum of exponentials for each row of the input matrix.
// This is used in a numerically stable softmax operation
// Each thread handles one row
//
// Parameters:
// - preActivations: Input matrix [rows x cols].
// - store_exp_sum: Output array [rows], stores the sum of exp(x - max) for each row.
// - store_max: Output array [rows], stores the max value in each row.
// - rows: Number of rows in the matrix.
// - cols: Number of columns in the matrix
__global__ void Kernels::calculate_max_and_sum_exp(float* preActivations,
	float* store_exp_sum, float* store_max, int rows, int cols){
	int row = blockDim.x * blockIdx.x + threadIdx.x;
	if (row < rows) {
		//Find max element in this row
		float max{ preActivations[row * cols]};
		float exp_sum{};
		for (int i {}; i<cols; ++i){
			int idx = row * cols + i;
			if (preActivations[idx] > max) max = preActivations[idx];
		}
		//Compute sum of exp(x - max) for numerical stability
		for (int i{}; i < cols; ++i) {
			int idx = row * cols + i;
			exp_sum += std::expf(preActivations[idx] - max);
		}
		store_max[row] = max;
		store_exp_sum[row] = exp_sum;
	}
}

// CUDA kernel to compute the softmax activation row-wise
// Uses precomputed max and sum of exponentials for numerical stability
//
// Parameters:
// - preActivations: Input matrix [rows x cols] (raw scores from last layer)
// - activations: Output matrix [rows x cols] (softmax probabilities)
// - exp_sum: Array [rows], holds sum of exp(x - max) per row (from previous kernel)
// - max: Array [rows], holds max(x) per row (from previous kernel)
// - rows: Number of rows in the matrix (batch size)
// - cols: Number of output classes (number of neurons in the final layer)
__global__ void Kernels::softmax(float* preActivations, float* activations, float* exp_sum, float* max, int rows, int cols) {
	int row = blockDim.y * blockIdx.y + threadIdx.y;
	int col = blockDim.x * blockIdx.x + threadIdx.x;
	int idx = row * cols + col;
	float eps = 1e-7f; // Small constant to avoid division by zero

	if (row < rows && col < cols) {
		// Subtract max for numerical stability
		// Then divide by sum of exponentials for proper normalization
		activations[idx] = std::expf(preActivations[idx] - max[row]) / (exp_sum[row]+eps);
	}
}

// CUDA kernel to perform element-wise multiplication of two matrices.
// Each element: store_in[i][j] = matrix_A[i][j] * matrix_B[i][j]
//
// Parameters:
// - matrix_A: Input matrix A [rows x cols]
// - matrix_B: Input matrix B [rows x cols]
// - store_in: Output matrix to store the result [rows x cols]
// - rows: Number of rows in the matrices
// - cols: Number of columns in the matrices
__global__ void Kernels::elementWiseProduct(float* matrix_A, float* matrix_B, float* store_in, int rows, int cols) {
	int row = blockDim.y * blockIdx.y + threadIdx.y; //Row Index
	int col = blockDim.x * blockIdx.x + threadIdx.x; //Col Index
	int idx = row * cols + col;

	if (row < rows && col < cols) {
		store_in[idx] = matrix_A[idx] * matrix_B[idx]; //Do Multiplication
	}
}

// CUDA kernel to compute the derivative (gradient) of activation functions.
// The result is stored in 'store_in' and used during backpropagation.
//
// Parameters:
// - preActivations: Input matrix (pre-activation values) [rows x cols]
// - store_in: Output matrix to store the computed gradients [rows x cols]
// - activation_id: Specifies which activation function to use:
//       0 = Sigmoid
//       1 = ReLU
//       2 = Leaky ReLU
//       3 = Tanh
// - rows: Number of rows in the matrices
// - cols: Number of columns in the matrices
__global__ void Kernels::activationPrime(float* preActivations, float* store_in, int activation_id, int rows, int cols) {
	int row = blockDim.y * blockIdx.y + threadIdx.y;
	int col = blockDim.x * blockIdx.x + threadIdx.x;
	int idx = row * cols + col;

	if (row < rows && col < cols) {
		float element = preActivations[idx];
		// Sigmoid derivative: σ'(x) = σ(x) * (1 - σ(x))
		if (activation_id == 0) {
			float sigmoid = 1.0f / (1.0f + std::expf(-element));
			store_in[idx] = sigmoid * (1.0f - sigmoid);
		}
		// ReLU derivative: 1 if x > 0, else 0
		else if (activation_id == 1) store_in[idx] = element > 0.0f ? 1.0f : 0.0f;
		// Leaky ReLU derivative: 1 if x > 0, else 0.1
		else if (activation_id == 2) store_in[idx] = element > 0.0f ? 1.0f : 0.1f;
		// Tanh derivative: 1 - tanh^2(x)
		else if (activation_id == 3) store_in[idx] = 1 - std::tanhf(element) * std::tanhf(element);
	}
}

// CUDA kernel to compute the sum of each column in a matrix
// The result is stored in 'store_in', where store_in[col] = sum of elements in column 'col'
//
// Parameters:
// - matrix: Input matrix of shape [rows x cols]
// - store_in: Output vector of size [cols], where each element stores the column sum
// - rows: Number of rows in the matrix
// - cols: Number of columns in the matrix
__global__ void Kernels::rowWiseSum(float* matrix, float* store_in, int rows, int cols) {
	int col = blockDim.x * blockIdx.x + threadIdx.x;
	if (col < cols) {
		float sum{};
		// Iterate over all rows for the given column
		for (int row{}; row < rows; ++row) {
			sum += matrix[row * cols + col];
		}
		// Store the sum of the column
		store_in[col] = sum;
	}
}

// CUDA kernel to compute the weighted sum for a layer in an MLP
// Each thread computes one element of the preActivation matrix
//
// Parameters:
// - input: Input matrix of shape [M x K]
// - weights_transposed: Transposed weights matrix of shape [K x N]
// - preActivations: Output matrix to store weighted sums, shape [M x N]
// - biases: Bias vector of shape [N]
// - M: Number of input samples (batch size)
// - N: Number of neurons in the current layer
// - K: Number of inputs to each neuron (input dimension)
__global__ void Kernels::computeWeightedSum(float* input, float* weights_transposed,
	float* preActivations, float* biases, int M, int N, int K) {

	int row = blockIdx.y * blockDim.y + threadIdx.y; // Current sample
	int col = blockIdx.x * blockDim.x + threadIdx.x;  // Current neuron

	// Bounds check
	if (row < M && col < N) {
		float sum = 0.0f;
		// Compute dot product: input[row] · weights_T[:, col]
		for (int k{}; k < K; ++k) {
			sum += input[row * K + k] * weights_transposed[k * N + col];
		}
		// Add bias and store result in preActivations
		preActivations[row * N + col] = sum + biases[col];
	}
}

// CUDA kernel to compute the derivative of the cost w.r.t. activations (dC/da)
// Supports MSE, BCE, and CE loss functions.
//
// Parameters:
// - activations: Output of the activation function (a), shape [rows x cols]
// - Y_train: Ground truth labels (y), shape [rows x cols]
// - errors: Output buffer to store dC/da, shape [rows x cols]
// - lossFn_id: ID representing the loss function used
//      0 -> MSE, 1 -> Binary Cross Entropy, 2 -> Cross Entropy
// - rows, cols: Dimensions of the input
__global__ void Kernels::compute_dC_da(float* activations, float* Y_train, float* errors,
	int lossFn_id, int rows, int cols) {

	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int idx = row * cols + col;

	// Bounds check
	if (row < rows && col < cols) {
		float y = Y_train[idx], a = activations[idx], eps = 1e-7f;
		// MSE loss: dC/da = a - y
		if (lossFn_id == 0) errors[idx] = a-y;
		// Binary Cross Entropy: dC/da = (a - y) / [(a + eps)(1 - a + eps)]
		else if (lossFn_id == 1) errors[idx] = (a - y) / ((a + eps) * (1.0f - a + eps));
		// Cross Entropy: dC/da = -y / (a + eps), Only valid if not using softmax activation
		else if (lossFn_id == 2) errors[idx] = -y / (a + eps);
	}

}

// CUDA kernel to compute total cross-entropy loss over a batch
// The result is accumulated into a single float using atomicAdd.
//
// Parameters:
// - activations: Predicted probabilities from softmax, shape [rows x cols]
// - Y_train: One-hot encoded true labels, shape [rows x cols]
// - loss: Pointer to a single float to store total loss (use hipMallocManaged or hipMemcpy to access on host)
// - rows: Number of samples (batch size)
// - cols: Number of classes (output neurons)
__global__ void Kernels::cross_entropy_loss(float* activations, float* Y_train, float* loss, int rows, int cols) {
	// Each thread processes one row (sample)
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	if (row < rows) {
		float local_loss{};
		for (int col{}; col < cols; ++col) {
			int idx = row * cols + col;
			// Compute y * log(a)
			local_loss += Y_train[idx] * std::logf(activations[idx]);
		}
		// Accumulate the negative of the local loss into the shared loss variable
		atomicAdd(loss, -local_loss);
	}

}

// CUDA kernel to compute total binary cross-entropy loss for a batch
// The result is accumulated into a single float using atomicAdd
//
// Parameters:
// - activations: Predicted probabilities (from sigmoid), shape [rows x cols]
// - Y_train: True binary labels, shape [rows x cols]
// - loss: Pointer to a float that stores total loss
// - rows: Number of samples
// - cols: Number of outputs per sample
__global__ void Kernels::binary_cross_entropy(float* activations, float* Y_train, float* loss, int rows, int cols) {
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	if (row < rows) {
		float local_loss{};
		for (int col{}; col < cols; ++col) {
			int idx = row * cols + col;
			const float eps = 1e-7f;
			float a{ activations[idx] }, y{Y_train[idx]};
			// BCE formula per element
			local_loss += y * std::logf(a+eps) + (1.0f - y) * std::logf(1.0f - a+eps);
		}
		// Add the loss from this row to the total loss atomically
		atomicAdd(loss, -local_loss);
	}
}

// CUDA kernel to compute total Mean Squared Error (MSE) loss
// Each thread computes the squared error for one row/sample.
//
// Parameters:
// - activations: Predicted outputs [rows x cols]
// - Y_train: Ground truth labels [rows x cols]
// - loss: Pointer to float that accumulates total MSE (to be averaged later)
// - rows: Number of data samples
// - cols: Number of output features per sample
__global__ void Kernels::MSE(float* activations, float* Y_train, float* loss, int rows, int cols) {
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	if (row < rows) {
		float local_loss{};
		for (int col{}; col < cols; ++col) {
			int idx = row * cols + col;
			float a{ activations[idx] }, y{ Y_train[idx] };
			local_loss += (y - a) * (y - a); // Square error for each output
		}
		// Add the loss from this row to the total loss atomically
		atomicAdd(loss, local_loss);
	}
}